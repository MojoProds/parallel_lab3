#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

const int TPB = 1024;

__global__ void getmaxcu(unsigned int* numbers_d, unsigned int* max_d, int n) {

	extern __shared__ unsigned int shared[];

	int tid = threadIdx.x;
	int gid = (blockDim.x * blockIdx.x) + tid;
	shared[tid] = 0;

	if (gid < n) {
		shared[tid] = numbers_d[gid];
	}
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s = s / 2) {
		if (tid < s && gid < n) {
			shared[tid] = max(shared[tid], shared[tid + s]);
		}
		__syncthreads();
	}

	if (tid == 0) {
		max_d[blockIdx.x] = shared[tid];
	}
}

void printArr(unsigned int num[], unsigned int size) {
	unsigned int i;

	for(i = 0; i < size; i++) {
		if(num[i] > 0) {
			printf("%u, ", num[i]);
			if(i % 7 == 0 || i == size - 1) {
				printf("\n");
			}
		}
	}
}

int main(int argc, char *argv[]) {
	unsigned int size = 0;  // The size of the array
	unsigned int i;  // loop index
	unsigned int * numbers; //pointer to the array
	
	if(argc !=2) {
	   printf("usage: maxseq num\n");
	   printf("num = size of the array\n");
	   exit(1);
	}
   
	size = atol(argv[1]);

	numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
	if( !numbers ) {
	   printf("Unable to allocate mem for an array of size %u\n", size);
	   exit(1);
	}    

	srand(time(NULL)); // setting a seed for the random number generator
	// Fill-up the array with random numbers from 0 to size-1 
	for( i = 0; i < size; i++) {
	   numbers[i] = rand()  % size;    
	}

	// Memory allocation in the device
	unsigned int* numbers_d;
	unsigned int* max_d;
	hipMalloc((void**)&numbers_d, size * sizeof(unsigned int));
	hipMalloc((void**)&max_d, (size/TPB + 1) * sizeof(unsigned int));

	// Call kernel
	int done = 0;
	for( i = size; i > 0 && done == 0;) {
		//printf("\nPre-Iteration: %u\n", i);
		//printArr(numbers, i);
		hipMemcpy(numbers_d, numbers, i * sizeof(unsigned int), hipMemcpyHostToDevice);
		getmaxcu<<<(int)ceil((float)i / TPB),TPB, TPB * sizeof(unsigned int)>>>(numbers_d, max_d, i);
		i = (int)ceil((float)i / TPB);
		hipMemcpy(numbers, max_d, i * sizeof(unsigned int), hipMemcpyDeviceToHost);
		if(i == 1) {
			done = 1;
		}
		//printArr(numbers, i);
		//rintf("\nPost-Iteration: %u\n", i);
		

	}

	// Print info
	printf("The maximum number in the array is: %u\n", numbers[0]);

	// Free memory
	hipFree(numbers_d);
	hipFree(max_d);
	free(numbers);
	exit(0);
}